#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-2025 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <legate/utilities/typedefs.h>

#include <cstddef>

namespace detail {

namespace {

__device__ __forceinline__ std::size_t global_tid_1d()
{
  return (static_cast<std::size_t>(blockIdx.x) * static_cast<std::size_t>(blockDim.x)) +
         static_cast<std::size_t>(threadIdx.x);
}

}  // namespace

}  // namespace detail

extern "C" __global__ void legate_ranges_to_offsets_kernel(
  std::size_t ranges_volume,
  legate::Point<1> ranges_lo,
  legate::AccessorWO<std::int32_t, 1> offsets_acc,    // NOLINT(performance-unnecessary-value-param)
  legate::AccessorRO<legate::Rect<1>, 1> ranges_acc)  // NOLINT(performance-unnecessary-value-param)
{
  if (const auto tid = detail::global_tid_1d(); tid < ranges_volume) {
    const auto p = ranges_lo + static_cast<legate::coord_t>(tid);

    offsets_acc[p] = static_cast<std::int32_t>(ranges_acc[p].lo[0] - ranges_acc[ranges_lo].lo[0]);
  }
}
