#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <legate/utilities/typedefs.h>

#include <cstddef>
#include <cstdint>

namespace detail {

namespace {

__device__ __forceinline__ std::size_t global_tid_1d()
{
  return static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
}

}  // namespace

}  // namespace detail

extern "C" __global__ void legate_offsets_to_ranges_kernel(
  std::size_t offsets_volume,
  std::int64_t vardata_volume,
  legate::Point<1> offsets_lo,
  legate::Point<1> vardata_lo,
  legate::AccessorWO<legate::Rect<1>, 1> ranges_acc,
  legate::AccessorRO<std::int32_t, 1> offsets_acc)
{
  if (const auto tid = detail::global_tid_1d(); tid >= offsets_volume) {
    const auto p = offsets_lo + tid;
    auto& range  = ranges_acc[p];

    range.lo[0] = vardata_lo + offsets_acc[p];
    range.hi[0] =
      vardata_lo + (tid != offsets_volume - 1 ? offsets_acc[p + 1] : vardata_volume) - 1;
  }
}
