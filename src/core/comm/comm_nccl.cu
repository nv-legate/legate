/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "core/comm/comm_nccl.h"
#include "core/cuda/cuda_help.h"
#include "core/cuda/stream_pool.h"
#include "core/data/buffer.h"
#include "core/utilities/nvtx_help.h"
#include "core/utilities/typedefs.h"
#include "legate.h"

#include <hip/hip_runtime.h>
#include <nccl.h>
#include <chrono>

namespace legate {
namespace comm {
namespace nccl {

struct _Payload {
  uint64_t field0;
  uint64_t field1;
};

#define CHECK_NCCL(expr)                    \
  do {                                      \
    ncclResult_t result = (expr);           \
    check_nccl(result, __FILE__, __LINE__); \
  } while (false)

inline void check_nccl(ncclResult_t error, const char* file, int line)
{
  if (error != ncclSuccess) {
    fprintf(stderr,
            "Internal NCCL failure with error %s in file %s at line %d\n",
            ncclGetErrorString(error),
            file,
            line);
    exit(error);
  }
}

static ncclUniqueId init_nccl_id(const Legion::Task* task,
                                 const std::vector<Legion::PhysicalRegion>& regions,
                                 Legion::Context context,
                                 Legion::Runtime* runtime)
{
  legate::nvtx::Range auto_range("core::comm::nccl::init_id");

  Core::show_progress(task, context, runtime);

  ncclUniqueId id;
  CHECK_NCCL(ncclGetUniqueId(&id));

  return id;
}

static ncclComm_t* init_nccl(const Legion::Task* task,
                             const std::vector<Legion::PhysicalRegion>& regions,
                             Legion::Context context,
                             Legion::Runtime* runtime)
{
  legate::nvtx::Range auto_range("core::comm::nccl::init");

  Core::show_progress(task, context, runtime);

  assert(task->futures.size() == 1);

  auto id          = task->futures[0].get_result<ncclUniqueId>();
  ncclComm_t* comm = new ncclComm_t{};

  auto num_ranks = task->index_domain.get_volume();
  auto rank_id   = task->index_point[0];

  auto ts_init_start = std::chrono::high_resolution_clock::now();
  CHECK_NCCL(ncclGroupStart());
  CHECK_NCCL(ncclCommInitRank(comm, num_ranks, id, rank_id));
  CHECK_NCCL(ncclGroupEnd());
  auto ts_init_stop = std::chrono::high_resolution_clock::now();

  auto time_init = std::chrono::duration<double>(ts_init_stop - ts_init_start).count() * 1000.0;

  if (0 == rank_id) { log_legate.debug("NCCL initialization took %lf ms", time_init); }

  if (num_ranks == 1) return comm;

  if (!Core::warmup_nccl) return comm;

  auto stream = cuda::StreamPool::get_stream_pool().get_stream();

  // Perform a warm-up all-to-all

  hipEvent_t ev_start, ev_end_all_to_all, ev_end_all_gather;
  CHECK_CUDA(hipEventCreate(&ev_start));
  CHECK_CUDA(hipEventCreate(&ev_end_all_to_all));
  CHECK_CUDA(hipEventCreate(&ev_end_all_gather));

  auto src_buffer = create_buffer<_Payload>(num_ranks, Memory::Kind::GPU_FB_MEM);
  auto tgt_buffer = create_buffer<_Payload>(num_ranks, Memory::Kind::GPU_FB_MEM);

  CHECK_CUDA(hipEventRecord(ev_start, stream));

  CHECK_NCCL(ncclGroupStart());
  for (auto idx = 0; idx < num_ranks; ++idx) {
    CHECK_NCCL(ncclSend(src_buffer.ptr(0), sizeof(_Payload), ncclInt8, idx, *comm, stream));
    CHECK_NCCL(ncclRecv(tgt_buffer.ptr(0), sizeof(_Payload), ncclInt8, idx, *comm, stream));
  }
  CHECK_NCCL(ncclGroupEnd());

  CHECK_CUDA(hipEventRecord(ev_end_all_to_all, stream));

  CHECK_NCCL(ncclAllGather(src_buffer.ptr(0), tgt_buffer.ptr(0), 1, ncclUint64, *comm, stream));

  CHECK_CUDA(hipEventRecord(ev_end_all_gather, stream));

  CHECK_CUDA(hipEventSynchronize(ev_end_all_gather));

  float time_all_to_all = 0.;
  float time_all_gather = 0.;
  CHECK_CUDA(hipEventElapsedTime(&time_all_to_all, ev_start, ev_end_all_to_all));
  CHECK_CUDA(hipEventElapsedTime(&time_all_gather, ev_end_all_to_all, ev_end_all_gather));

  if (0 == rank_id) {
    log_legate.debug("NCCL warm-up took %f ms (all-to-all: %f ms, all-gather: %f ms)",
                     time_all_to_all + time_all_gather,
                     time_all_to_all,
                     time_all_gather);
  }

  return comm;
}

static void finalize_nccl(const Legion::Task* task,
                          const std::vector<Legion::PhysicalRegion>& regions,
                          Legion::Context context,
                          Legion::Runtime* runtime)
{
  legate::nvtx::Range auto_range("core::comm::nccl::finalize");

  Core::show_progress(task, context, runtime);

  assert(task->futures.size() == 1);
  auto comm = task->futures[0].get_result<ncclComm_t*>();
  CHECK_NCCL(ncclCommDestroy(*comm));
  delete comm;
}

void register_tasks(Legion::Machine machine,
                    Legion::Runtime* runtime,
                    const LibraryContext* context)
{
  auto init_nccl_id_task_id          = context->get_task_id(LEGATE_CORE_INIT_NCCL_ID_TASK_ID);
  const char* init_nccl_id_task_name = "core::comm::nccl::init_id";
  runtime->attach_name(
    init_nccl_id_task_id, init_nccl_id_task_name, false /*mutable*/, true /*local only*/);

  auto init_nccl_task_id          = context->get_task_id(LEGATE_CORE_INIT_NCCL_TASK_ID);
  const char* init_nccl_task_name = "core::comm::nccl::init";
  runtime->attach_name(
    init_nccl_task_id, init_nccl_task_name, false /*mutable*/, true /*local only*/);

  auto finalize_nccl_task_id          = context->get_task_id(LEGATE_CORE_FINALIZE_NCCL_TASK_ID);
  const char* finalize_nccl_task_name = "core::comm::nccl::finalize";
  runtime->attach_name(
    finalize_nccl_task_id, finalize_nccl_task_name, false /*mutable*/, true /*local only*/);

  auto make_registrar = [&](auto task_id, auto* task_name, auto proc_kind) {
    Legion::TaskVariantRegistrar registrar(task_id, task_name);
    registrar.add_constraint(Legion::ProcessorConstraint(proc_kind));
    registrar.set_leaf(true);
    registrar.global_registration = false;
    return registrar;
  };

  // Register the task variants
  {
    auto registrar =
      make_registrar(init_nccl_id_task_id, init_nccl_id_task_name, Processor::TOC_PROC);
    runtime->register_task_variant<ncclUniqueId, init_nccl_id>(registrar, LEGATE_GPU_VARIANT);
  }
  {
    auto registrar = make_registrar(init_nccl_task_id, init_nccl_task_name, Processor::TOC_PROC);
    runtime->register_task_variant<ncclComm_t*, init_nccl>(registrar, LEGATE_GPU_VARIANT);
  }
  {
    auto registrar =
      make_registrar(finalize_nccl_task_id, finalize_nccl_task_name, Processor::TOC_PROC);
    runtime->register_task_variant<finalize_nccl>(registrar, LEGATE_GPU_VARIANT);
  }
}

bool needs_barrier()
{
  // Blocking communications in NCCL violate CUDA's (undocumented) concurrent forward progress
  // requirements and no CUDA drivers that have released are safe from this. Until either CUDA
  // or NCCL is fixed, we will always insert a barrier at the beginning of every NCCL task.
  return true;
}

}  // namespace nccl
}  // namespace comm
}  // namespace legate
